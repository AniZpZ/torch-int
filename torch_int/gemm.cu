#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <torch/torch.h>


torch::Tensor igemm(torch::Tensor A, torch::Tensor B)
 {
    // Input: A, B are 2D signed 8-bit integer tensors
    // Output: C is 2D signed 32-bit integer tensor
    // B is transposed
    
    const int alpha = 1;
    const int beta = 0; // C = alpha * A * B + beta * C


    int m = A.size(0);
    int k = A.size(1);
    int n = B.size(0);
    
    hipblasStatus_t stat;
    
    hipDataType Atype = HIP_R_8I;
    hipDataType Btype = HIP_R_8I;
    hipDataType Ctype = HIP_R_32I;
    hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32I;
    hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;

    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS initialization failed" << std::endl;
    }
    
    // hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    
    torch::Tensor C = torch::zeros({m, n}, torch::dtype(torch::kInt).device(torch::kCUDA));

    stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A.data_ptr(), Atype, m, B.data_ptr(), Btype, k, &beta, C.data_ptr(), Ctype, m, computeType, algo);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("CUBLAS GEMM failed, error code: " + std::string(cublasGetStatusString(stat)));
    }

    return C;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("igemm", &igemm, "igemm (CUDA)");
}