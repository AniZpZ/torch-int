#include "include/gemm.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <torch/torch.h>

#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/numeric_types.h>
#include <cutlass/util/host_tensor.h>

torch::Tensor gemm_cutlass(torch::Tensor input, torch::Tensor weight) {

  auto M = input.size(0);
  auto N = weight.size(0);
  auto K = input.size(1);

  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementComputeEpilogue = float;
  using ElementInputA = int8_t; // <- data type of elements in input matrix A
  using ElementInputB = int8_t; // <- data type of elements in input matrix B

  // The code section below describes matrix layout of input and output
  // matrices. Column Major for Matrix A, Row Major for Matrix B and Row Major
  // for Matrix C
  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  using Gemm = cutlass::gemm::device::Gemm<
      int8_t, cutlass::layout::RowMajor, int8_t, cutlass::layout::ColumnMajor,
      ElementOutput, cutlass::layout::RowMajor, ElementAccumulator,
      cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<256, 128, 64>,
      cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<8, 8, 16>,
      cutlass::epilogue::thread::LinearCombinationClamp<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementComputeEpilogue>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3>;

  auto input_size = cutlass::MatrixCoord(M, K);
  auto weight_size = cutlass::MatrixCoord(K, N);
  auto output_size = cutlass::MatrixCoord(M, N);

  auto device = input.device();
  auto out = torch::empty({M, N}, torch::dtype(torch::kInt32).device(device));

  // constexpr int kSparse = Gemm::kSparse;
  // How many elements of A are covered per ElementE
  // constexpr int kElementsPerElementE = Gemm::kElementsPerElementE;
  // The size of individual meta data
  // constexpr int kMetaSizeInBits = Gemm::kMetaSizeInBits;
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  cutlass::TensorRef<ElementInputA, LayoutInputA> input_ref(
      input.data_ptr<int8_t>(), LayoutInputA::packed(input_size));
  cutlass::TensorRef<ElementInputB, LayoutInputB> weight_ref(
      weight.data_ptr<int8_t>(), LayoutInputB::packed(weight_size));
  cutlass::TensorRef<ElementOutput, LayoutOutput> out_ref(
      out.data_ptr<int32_t>(), LayoutOutput::packed(output_size));

  // Initialize alpha and beta for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);
  ElementComputeEpilogue beta = ElementComputeEpilogue(0);

  typename Gemm::Arguments arguments{
      problem_size, // <- problem size of matrix multiplication
      input_ref,    // <- reference to matrix A on device
      weight_ref,   // <- reference to matrix B on device
      out_ref,            // <- reference to matrix C on device
      out_ref,   // <- reference to matrix D on device
      {alpha, beta},
      1};
  Gemm gemm_op;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = gemm_op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass gemm failed, error code");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass gemm failed, error code");
  }

  status = gemm_op();
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass gemm failed, error code");
  }

  return out;
}
