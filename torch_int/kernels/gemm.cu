#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <torch/extension.h>
#include <torch/torch.h>

torch::Tensor gemm(torch::Tensor A, torch::Tensor B, bool transa, bool transb) {
  // Input: A, B are 2D signed 8-bit integer tensors
  // Output: C is 2D signed 32-bit integer tensor

  const int alpha = 1;
  const int beta = 0; // C = alpha * A * B + beta * C

  hipblasStatus_t stat;

  hipDataType Atype = HIP_R_8I;
  hipDataType Btype = HIP_R_8I;
  hipDataType Ctype = HIP_R_32I;
  hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32I;
  hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;

  hipblasHandle_t handle;

  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    throw std::runtime_error("CUBLAS initialization failed");
  }

  torch::Device device = A.device();

  hipblasOperation_t transa_ = transa ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transb_ = transb ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  int n = transa ? A.size(1) : A.size(0);
  int k = transa ? A.size(0) : A.size(1);
  int m = transb ? B.size(0) : B.size(1);

  int lda = transa ? n : k;
  int ldb = transb ? k : m;
  int ldc = m;

  torch::Tensor C =
      torch::zeros({n, m}, torch::dtype(torch::kInt).device(device));

  // Note that CuBLAS assumes column-major matrices, so we actually pass the
  // transposes of A and B
  stat = hipblasGemmEx(handle, transb_, transa_, m, n, k, &alpha, B.data_ptr(),
                      Btype, ldb, A.data_ptr(), Atype, lda, &beta, C.data_ptr(),
                      Ctype, ldc, computeType, algo);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    throw std::runtime_error("CUBLAS GEMM failed, error code: " +
                             std::string(cublasGetStatusString(stat)));
  }

  return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("gemm", &gemm, "GEMM (CUDA)");
}